#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "PointsGenerator.h"
#include "Stopwatch.h"

#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <math.h>
#include <cmath>
#include <thrust/count.h>
#include <thrust/fill.h>

const unsigned long N = 1000000UL;
//const unsigned int dim = 3;
const double THRESHOLD = 0.01;
const unsigned int K = 5;
const bool SHOULD_DEBUG = false;
const int SHARED_MEM_SIZE = 24576;
const int MEMBERSHIP_SHARED_MEM_SIZE = SHARED_MEM_SIZE / sizeof(short);

template <unsigned int dim>
void displayPointsDevice(const thrust::host_vector<float>& points)
{
    unsigned long pointsCount = points.size() / dim;
    for (size_t i = 0; i < pointsCount; i++)
    {
        for (size_t j = 0; j < dim; j++)
        {
            std::cout << std::setw(9) << std::setprecision(6) << points[i + j * pointsCount];
        }
        std::cout << std::endl;
    }
}

template <unsigned int dim>
void displayPointsHost(const std::vector<std::vector<float>>& points)
{
    for (size_t i = 0; i < points.size(); i++)
    {
        for (size_t j = 0; j < dim; j++)
        {
            std::cout << std::setw(9) << std::setprecision(6) << points[i][j];
        }
        std::cout << std::endl;
    }
}

template <unsigned int dim>
struct Point
{
    const float* start_ptr;
    unsigned long stride;
    __device__ Point(const float* start, const unsigned long stride)
    {
        this->start_ptr = start;
        this->stride = stride;
    }
    __device__ float operator [] (unsigned long index) const
    {
        return start_ptr[index * stride];
    }
};

template<int dim>
__device__ float distance(const Point<dim>& p1, const Point<dim>& p2)
{
    float sum = 0.0f;
    for (size_t i = 0; i < dim; i++)
        sum += pow(p1[i] - p2[i], 2.0f);

    return sqrt(sum);
}

template<int dim>
__device__ short findNewCentroid(const Point<dim>& point, const float* centroids)
{
    float minDist = INT_MAX, tempDist;
    short centroidIndex = -1;
    for (short j = 0; j < K; j++)
    {
        tempDist = distance(point, Point<dim>(centroids + j, K));
        if (minDist > tempDist)
        {
            minDist = tempDist;
            centroidIndex = j;
        }
    }

    return centroidIndex;
}

template<int dim>
__global__ void assignPointToCentroid(const float *points, const float *centroids, short* membership, short* delta)
{
    int threadIndex = threadIdx.x + blockIdx.x * blockDim.x;
    if (threadIndex > N)
        return;

    Point<dim> point{ points + threadIndex, N };

    short centroidIndex = findNewCentroid(point, centroids);

    if (membership[threadIndex] != centroidIndex)
    {
        membership[threadIndex] = centroidIndex;
        delta[threadIndex] = 1;
    }
}

template<int dim>
__global__ void assignPointToCentroidWithSegmentation(const float* points, const float* centroids, short* membership, float *output, short* delta)
{
    int threadIndex = threadIdx.x + blockIdx.x * blockDim.x;
    if (threadIndex > N)
        return;

    Point<dim> point{ points + threadIndex, N };

    short prevCentroid = membership[threadIndex];
    short newCentroid = findNewCentroid(point, centroids);

    if (prevCentroid != newCentroid)
    {
        if (prevCentroid == -1)
            for (size_t i = 0; i < dim; i++)
                output[threadIndex + N * (dim * newCentroid + i)] = point[i];
        else
            for (size_t i = 0; i < dim; i++)
            {
                output[threadIndex + N * (dim * prevCentroid + i)] = 0.0f;
                output[threadIndex + N * (dim * newCentroid + i)] = point[i];
            }

        membership[threadIndex] = newCentroid;
        delta[threadIndex] = 1;
    }
}

template<int dim>
__global__ void findNewCentroids(const float* points, const short* membership, float * newCentroids, const unsigned long * centroidsSizes)
{
    __shared__ short local_membership[MEMBERSHIP_SHARED_MEM_SIZE];
    int threadIndex = threadIdx.x;
    int centroidIndex = threadIndex / dim;
    unsigned long stride = threadIndex - dim * centroidIndex;
    unsigned long size = centroidsSizes[centroidIndex];
    unsigned long sharedMemStride = blockDim.x;
    float center = 0;

    size_t index = 0;
    while (index < N)
    {
        for (size_t i = threadIndex; i < MEMBERSHIP_SHARED_MEM_SIZE && i + index < N; i += sharedMemStride)
            local_membership[i] = membership[index + i];
        __syncthreads();

        for (size_t i = 0; i < MEMBERSHIP_SHARED_MEM_SIZE && i + index < N; i++, index++)
            if (local_membership[i] == centroidIndex)
                center += points[index + N * stride] / size;
    }

    newCentroids[stride * K + centroidIndex] = center;
}

template<int dim>
void lloyd_gpu_kernel(const thrust::device_vector<float>& points, thrust::device_vector<float> centroids)
{
    thrust::device_vector<short> membership{ N, 0 };
    thrust::device_vector<short> delta{ N, 0 };
    thrust::device_vector<unsigned long> centroidsSizes{ K };

    unsigned long deltaCount = N;
    int threads = 1024;
    int blocks = ceil(N / (double)threads);
    float time;

    hipError_t err;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    while (deltaCount / (double)N > THRESHOLD)
    {
        if (SHOULD_DEBUG) 
        {
            std::cout << "Centroids: " << std::endl;
            displayPointsDevice<dim>(centroids);
        }

        assignPointToCentroid<dim> <<< blocks, threads >>> (thrust::raw_pointer_cast(points.data()), thrust::raw_pointer_cast(centroids.data()), thrust::raw_pointer_cast(membership.data()), thrust::raw_pointer_cast(delta.data()));
        err = hipGetLastError();
        if (err != hipSuccess) printf("%s\n", hipGetErrorString(err));


        for (size_t i = 0; i < K; i++)
            centroidsSizes[i] = thrust::count(membership.begin(), membership.end(), i);

        findNewCentroids<dim> <<<1, dim*K>>> (thrust::raw_pointer_cast(points.data()), thrust::raw_pointer_cast(membership.data()), 
                                         thrust::raw_pointer_cast(centroids.data()), thrust::raw_pointer_cast(centroidsSizes.data()));
        

        deltaCount = thrust::count(delta.begin(), delta.end(), 1);
        thrust::fill(delta.begin(), delta.end(), 0);


        std::cout << "delta: " << deltaCount / (double)N << std::endl;
        if (SHOULD_DEBUG)
        {
            for (size_t i = 0; i < K ; i++)
                std::cout << centroidsSizes[i] << " ";
            std::cout << std::endl << std::endl;
            std::cout << "Membership:" << std::endl;
            for (size_t i = 0; i < N; i++)
                std::cout << membership[i] << " ";
            std::cout << std::endl;
        }
        
    }
    hipEventRecord(stop, 0);
    hipDeviceSynchronize();
    hipEventElapsedTime(&time, start, stop);
    displayPointsDevice<dim>(centroids);
    std::cout << "GPU-Kernel time: " << time << " ms" << std::endl;
}

template<int dim>
void lloyd_gpu_reduce(const thrust::device_vector<float>& points, thrust::device_vector<float> centroids)
{
    thrust::device_vector<short> membership{ N, -1 };
    thrust::device_vector<short> delta{ N, 0 };
    thrust::device_vector<unsigned long> centroidsSizes{ K };
    thrust::device_vector<float> output{N * dim * K, 0};

    int threads = 1024;
    int blocks = ceil(N / (double)threads);
    unsigned long deltaCount = N;

    float time;
    hipError_t err;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    while (deltaCount / (double)N > THRESHOLD)
    {
        if (SHOULD_DEBUG)
        {
            std::cout << "Centroids: " << std::endl;
            displayPointsDevice<dim>(centroids);
        }
        assignPointToCentroidWithSegmentation<dim> <<< blocks, threads >>> (thrust::raw_pointer_cast(points.data()), thrust::raw_pointer_cast(centroids.data()), 
                                                                       thrust::raw_pointer_cast(membership.data()), thrust::raw_pointer_cast(output.data()),
                                                                       thrust::raw_pointer_cast(delta.data()));
        err = hipGetLastError();
        if (err != hipSuccess) printf("%s\n", hipGetErrorString(err));


        for (size_t i = 0; i < K; i++)
            centroidsSizes[i] = thrust::count(membership.begin(), membership.end(), i);

        for (size_t centroid = 0; centroid < K; centroid++)
            for (size_t dimension = 0; dimension < dim; dimension++)
                centroids[centroid + dimension * K] = thrust::reduce(output.begin() + N * (dim * centroid + dimension), output.begin() + N * (dim * centroid + dimension + 1)) / centroidsSizes[centroid];
        

        deltaCount = thrust::count(delta.begin(), delta.end(), 1);
        thrust::fill(delta.begin(), delta.end(), 0);


        std::cout << "delta: " << deltaCount / (double)N << std::endl;
        if (SHOULD_DEBUG)
        {
            for (size_t i = 0; i < K; i++)
                std::cout << centroidsSizes[i] << " ";
            std::cout << std::endl << std::endl;
            std::cout << "Membership:" << std::endl;
            for (size_t i = 0; i < N; i++)
                std::cout << membership[i] << " ";
            std::cout << std::endl;
        }
    }
    hipEventRecord(stop, 0);
    hipDeviceSynchronize();
    hipEventElapsedTime(&time, start, stop);
    displayPointsDevice<dim>(centroids);
    std::cout << "GPU-Reduce time: " << time << " ms" << std::endl;
}

// Calcuates the Euclidean distance between two points
template <unsigned int dim>
float distance(const std::vector<float>& p1, const std::vector<float>& p2)
{
    double sum = 0;
    for (size_t i = 0; i < dim; i++)
        sum += pow(double(p2[i] - p1[i]), 2.0);

    return sqrt(sum);
}

template <unsigned int dim>
void lloyd_cpu(const std::vector<std::vector<float>>& points, std::vector<std::vector<float>> centroids)
{
    unsigned long delta = N;
    short* membership = new short[N] {};
    unsigned long centroidsSizes[K] {};
    float newCentroids[K][dim] {};

    if (SHOULD_DEBUG) displayPointsHost<dim>(centroids);

    while (delta / (float)N > THRESHOLD)
    {
        delta = 0;
        for (size_t i = 0; i < N; i++)
        {
            float minDist = INT_MAX, tempDist;
            short index = -1;
            for (short j = 0; j < K; j++)
            {
                tempDist = distance<dim>(points[i], centroids[j]);
                if (minDist > tempDist)
                {
                    minDist = tempDist;
                    index = j;
                }
            }
            centroidsSizes[index]++;
            for (size_t j = 0; j < dim; j++)
                newCentroids[index][j] += points[i][j];

            if (membership[i] != index)
            {
                membership[i] = index;
                delta++;
            }
        }

        std::cout << "delta: " << delta / (double)N << std::endl;
        if (SHOULD_DEBUG) displayPointsHost<dim>(centroids);

        for (size_t i = 0; i < K; i++)
        {
            for (size_t j = 0; j < dim; j++)
            {
                centroids[i][j] = newCentroids[i][j] / centroidsSizes[i];
                newCentroids[i][j] = 0;
            }
            centroidsSizes[i] = 0;
        }
     }

    if (SHOULD_DEBUG)
    {
        std::cout << "Membership:" << std::endl;
        for (size_t i = 0; i < N; i++)
            std::cout << membership[i] << " ";
        std::cout << std::endl;
    }
    displayPointsHost<dim>(centroids);

    delete[] membership;
}


int main()
{
    unsigned const int dim = 3;
    Stopwatch stopwatch;
    PointsGenerator<dim> gen;
    
    std::cout << "Generating points ";
    stopwatch.Start();
    auto d_points = gen.generatePointsDevice(N);
    stopwatch.Stop();

    std::cout << "Generating centroids ";
    stopwatch.Start();
    auto d_centroids = gen.generateCentroidsDevice(K);
    stopwatch.Stop();

    std::cout << "Copying points ";
    stopwatch.Start();
    auto h_points = gen.deviceToHost(d_points);
    auto h_centroids = gen.deviceToHost(d_centroids);
    stopwatch.Stop();

    std::cout << std::endl << "LLoyd CPU ";
    stopwatch.Start();
    lloyd_cpu<dim>(h_points, h_centroids);
    stopwatch.Stop();

    std::cout << std::endl << "LLoyd GPU Kernel" << std::endl;
    lloyd_gpu_kernel<dim>(d_points, d_centroids);

    std::cout << std::endl << "LLoyd GPU Reduce" << std::endl;
    lloyd_gpu_reduce<dim>(d_points, d_centroids);

    return 0;
}
