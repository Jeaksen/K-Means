#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "PointsGenerator.h"
#include "Stopwatch.h"

#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <math.h>
#include <cmath>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/count.h>
#include <thrust/fill.h>

const unsigned long N = 1000000UL;
const unsigned int DIM = 3;
const double THRESHOLD = 0.01;
const unsigned int K = 5;
const bool SHOULD_DEBUG = false;
const int SHARED_MEM_SIZE = 49152;
const int MEMBERSHIP_SHARED_MEM_SIZE = SHARED_MEM_SIZE / sizeof(short);

void displayPointsDevice(const thrust::host_vector<float>& points)
{
    unsigned long pointsCount = points.size() / DIM;
    for (size_t i = 0; i < pointsCount; i++)
    {
        for (size_t j = 0; j < DIM; j++)
        {
            std::cout << std::setw(9) << std::setprecision(6) << points[i + j * pointsCount];
        }
        std::cout << std::endl;
    }
}

void displayPointsHost(const std::vector<std::vector<float>>& points)
{
    for (size_t i = 0; i < points.size(); i++)
    {
        for (size_t j = 0; j < points[0].size(); j++)
        {
            std::cout << std::setw(9) << std::setprecision(6) << points[i][j];
        }
        std::cout << std::endl;
    }
}

template <unsigned int dim>
struct Point
{
    const float* start_ptr;
    unsigned long stride;
    __device__ Point(const float* start, const unsigned long stride)
    {
        this->start_ptr = start;
        this->stride = stride;
    }
    __device__ float operator [] (unsigned long index) const
    {
        return start_ptr[index * stride];
    }
};


__device__ float distance(const Point<DIM>& p1, const Point<DIM>& p2)
{
    float sum = 0.0f;
    for (size_t i = 0; i < DIM; i++)
        sum += pow(p1[i] - p2[i], 2.0f);

    return sqrt(sum);
}


__global__ void assignPointToCentroid(const float *points, const float *centroids, short* membership, short* delta)
{
    int threadIndex = threadIdx.x + blockIdx.x * blockDim.x;
    if (threadIndex > N)
        return;
    Point<DIM> point{ points + threadIndex, N };

    float minDist = INT_MAX, tempDist;
    short centroidIndex = -1;
    for (short j = 0; j < K; j++)
    {
        tempDist = distance(point, Point<DIM>(centroids + j, K));
        if (minDist > tempDist)
        {
            minDist = tempDist;
            centroidIndex = j;
        }
    }

    if (membership[threadIndex] != centroidIndex)
    {
        membership[threadIndex] = centroidIndex;
        delta[threadIndex] = 1;
    }
}

__global__ void findNewCentroids(const float* points, const short* membership, float * newCentroids, const unsigned long * centroidsSizes)
{
    int threadIndex = threadIdx.x;
    int centroidIndex = threadIndex / DIM;
    unsigned long stride = threadIndex - DIM * centroidIndex;
    unsigned long size = centroidsSizes[centroidIndex];
    float center = 0;

    for (size_t i = 0; i < N; i++)
    {
        if (membership[i] == centroidIndex)
            center += points[ i + N * stride] / size;
    }
    //printf("index: %d  centroid: %d  stride: %d  size: %d  center: %f  output index: %d\n", threadIndex, centroidIndex, stride, size, center, stride * K + centroidIndex);

    newCentroids[stride * K + centroidIndex] = center;
}


// Calcuates the Euclidean distance between two points
float distance(const std::vector<float>& p1, const std::vector<float>& p2)
{
    double sum = 0;
    for (size_t i = 0; i < DIM; i++)
        sum += pow(double(p2[i] - p1[i]), 2.0);

    return sqrt(sum);
}

void lloyd_gpu(const thrust::device_vector<float>& points, thrust::device_vector<float> centroids)
{
    thrust::device_vector<short> membership{ N, 0 };
    thrust::device_vector<short> delta{ N, 0 };
    thrust::device_vector<unsigned long> centroidsSizes{ K };
    thrust::constant_iterator<unsigned long> ones(1);
    unsigned long deltaCount = N;
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipError_t err;
    int threads = 1024;
    int blocks = ceil(N / (double)threads);
    hipEventRecord(start, 0);

    while (deltaCount / (double)N > THRESHOLD)
    {
        if (SHOULD_DEBUG) 
        {
            std::cout << "Centroids: " << std::endl;
            displayPointsDevice(centroids);
        }

        assignPointToCentroid <<< blocks, threads >>> (thrust::raw_pointer_cast(points.data()), thrust::raw_pointer_cast(centroids.data()), thrust::raw_pointer_cast(membership.data()), thrust::raw_pointer_cast(delta.data()));
        err = hipGetLastError();
        if (err != hipSuccess) printf("%s\n", hipGetErrorString(err));
        
        for (size_t i = 0; i < K; i++)
            centroidsSizes[i] = thrust::count(membership.begin(), membership.end(), i);

        findNewCentroids <<<1, DIM*K>>> (thrust::raw_pointer_cast(points.data()), thrust::raw_pointer_cast(membership.data()), thrust::raw_pointer_cast(centroids.data()), thrust::raw_pointer_cast(centroidsSizes.data()));

        deltaCount = thrust::count(delta.begin(), delta.end(), 1);
        thrust::fill(delta.begin(), delta.end(), 0);

        std::cout << "delta: " << deltaCount / (double)N << std::endl;
        if (SHOULD_DEBUG)
        {
            for (size_t i = 0; i < K ; i++)
                std::cout << centroidsSizes[i] << " ";
            std::cout << std::endl << std::endl;
            std::cout << "Membership:" << std::endl;
            for (size_t i = 0; i < N; i++)
                std::cout << membership[i] << " ";
            std::cout << std::endl;
        }
        
    }
    hipEventRecord(stop, 0);
    hipDeviceSynchronize();
    hipEventElapsedTime(&time, start, stop);
    std::cout << "GPU time: " << time << " ms" << std::endl;
    displayPointsDevice(centroids);

}

void lloyd_cpu(const std::vector<std::vector<float>>& points, std::vector<std::vector<float>> centroids)
{
    unsigned long delta = N;
    short* membership = new short[N] {};
    unsigned long centroidsSizes[K] {};
    float newCentroids[K][DIM] {};

    if (SHOULD_DEBUG) displayPointsHost(centroids);

    while (delta / (float)N > THRESHOLD)
    {
        delta = 0;
        for (size_t i = 0; i < N; i++)
        {
            float minDist = INT_MAX, tempDist;
            short index = -1;
            for (short j = 0; j < K; j++)
            {
                tempDist = distance(points[i], centroids[j]);
                if (minDist > tempDist)
                {
                    minDist = tempDist;
                    index = j;
                }
            }
            centroidsSizes[index]++;
            for (size_t j = 0; j < DIM; j++)
                newCentroids[index][j] += points[i][j];

            if (membership[i] != index)
            {
                membership[i] = index;
                delta++;
            }
        }

        std::cout << "delta: " << delta / (double)N << std::endl;
        if (SHOULD_DEBUG) displayPointsHost(centroids);

        for (size_t i = 0; i < K; i++)
        {
            for (size_t j = 0; j < DIM; j++)
            {
                centroids[i][j] = newCentroids[i][j] / centroidsSizes[i];
                newCentroids[i][j] = 0;
            }
            centroidsSizes[i] = 0;
        }
     }

    if (SHOULD_DEBUG)
    {
        std::cout << "Membership:" << std::endl;
        for (size_t i = 0; i < N; i++)
            std::cout << membership[i] << " ";
        std::cout << std::endl;
    }
    displayPointsHost(centroids);

    delete[] membership;
}



int main()
{
    Stopwatch stopwatch;
    PointsGenerator<DIM> gen;

    std::cout << "Generating points ";
    stopwatch.Start();
    auto d_points = gen.generatePointsDevice(N);
    stopwatch.Stop();

    auto d_centroids = gen.generateCentroidsDevice(K);
    //if (SHOULD_DEBUG) displayPointsHost(h_points);
        //std::cout << "Copying points ";
    //stopwatch.Start();
    auto h_points = gen.deviceToHost(d_points);
    auto h_centroids = gen.deviceToHost(d_centroids);
    //stopwatch.Stop();

    std::cout << std::endl << "LLoyd CPU ";
    stopwatch.Start();
    lloyd_cpu(h_points, h_centroids);
    stopwatch.Stop();

    std::cout << std::endl << "LLoyd GPU" << std::endl << std::endl;
    lloyd_gpu(d_points, d_centroids);


    return 0;
}
